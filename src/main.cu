//https://devblogs.nvidia.com/easy-introduction-cuda-c-and-c/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int runtimeVersion = -1;
    hipError_t error_id = hipRuntimeGetVersion(&runtimeVersion);

    printf("Runtime version %d; Cuda error: %x (%s)\n", runtimeVersion, error_id, hipGetErrorString(error_id));

    int driverVersion = -1;
    error_id = hipDriverGetVersion(&driverVersion);

    printf("Driver version %d; Cuda error: %x (%s)\n", driverVersion,error_id, hipGetErrorString(error_id));

    int deviceCount = -1;
    error_id = hipGetDeviceCount(&deviceCount);

    printf("Device count %d; Cuda error: %x (%s)\n", deviceCount, error_id, hipGetErrorString(error_id));
}
